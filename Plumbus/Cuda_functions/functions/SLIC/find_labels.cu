#include "hip/hip_runtime.h"
#include"../../../cuda_includes.h"
#include"../../../config.h"









__global__ void find_labels_kernel(cv::cuda::PtrStepSz<cv::Vec3b> src, cv::cuda::PtrStepSzi labels, cv::cuda::PtrStepSzi row_vals, cv::cuda::PtrStepSzi col_vals, cv::cuda::PtrStepSzi sector_LUT, int density, int k_step ) {

	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (row < 0|| col < 0 || row >= src.rows || col >= src.cols) { return; }

	int sector_rows = row_vals.rows;
	int sector_cols = row_vals.cols;


	int sector_row = (row - (row % k_step))/k_step;
	int sector_col = (col - (col % k_step)) / k_step;;
	int sector_id = (sector_row * sector_cols) + sector_col;
	int closest_centers[9][2];

	for (int center = 0; center < 9; center++) {

		int center_neighbor_sector_row_id = (sector_id * 9 * 2);
		int center_neighbor_sector_col_id = (sector_id * 9 * 2) + 1;

		closest_centers[center][0] = sector_LUT(0, center_neighbor_sector_row_id);
		closest_centers[center][1] = sector_LUT(0, center_neighbor_sector_col_id);
	}

	int closest_centers_actual[9][2];
	for (int center = 0; center < 9; center++) {
		int center_sector_row = closest_centers[center][0];
		int center_sector_col = closest_centers[center][1];

		int center_actual_row = -1;
		int center_actual_col = -1;

		if (center_sector_row != -1 || center_sector_col != -1) {
			int center_actual_row = row_vals(center_sector_row, center_sector_col);
			int center_actual_col = col_vals(center_sector_row, center_sector_col);
		}

		closest_centers_actual[center][0] = center_actual_row;
		closest_centers_actual[center][1] = center_actual_col;
	}


//based on sector, get sectors to check (look up table)

	int focus_L = src(row, col)[0];
	int focus_A = src(row, col)[1];
	int focus_B = src(row, col)[2];
	int focus_color[3] = { focus_L, focus_A, focus_B };

	int closest_center_id = -1;
	int closest_center_distance = INF;


//check all valid centers, determine which is nearest
	for (int center = 0; center < 9; center++) {

		int center_sector_row = closest_centers[center][0];
		int center_sector_col = closest_centers[center][1];

		if (center_sector_row == -1 || center_sector_col == -1) { break; }

		int center_id = (center_sector_row * sector_cols) + center_sector_col;

		int center_actual_row = row_vals(center_sector_row, center_sector_col);
		int center_actual_col = col_vals(center_sector_row, center_sector_col);

		int center_L = src(center_actual_row, center_actual_col)[0];
		int center_A = src(center_actual_row, center_actual_col)[0];
		int center_B = src(center_actual_row, center_actual_col)[0];
		int center_color[3] = { center_L, center_A, center_B };

		//perform distance check
		int channel_diff_sum = 0;
		for (int channel = 0; channel < 3; channel++) {
			int channel_diff = focus_color[channel] - center_color[channel];
			channel_diff_sum += channel_diff * channel_diff;
		}

		int dlab = sqrt(channel_diff_sum);
		int dxy = sqrt(((row - center_actual_row)*(row - center_actual_row)) + ((col - center_actual_col)*(col - center_actual_col)));
		int xy_mod = density/k_step;
		int distance_to_center = dlab + (xy_mod * dxy);

		if (distance_to_center < closest_center_distance) {
			closest_center_distance = distance_to_center;
			closest_center_id = center_id;
		}
	}

	labels(row, col) = closest_center_id;
}

















void find_labels_launch(cv::cuda::GpuMat& src, cv::cuda::GpuMat& labels, cv::cuda::GpuMat& row_vals, cv::cuda::GpuMat& col_vals, cv::cuda::GpuMat& sector_LUT, int density, int k_step) {



	int rows = row_vals.rows;
	int cols = row_vals.cols;

	int K = rows * cols;

	unsigned int block_dim_xy = 32;
	unsigned int grid_dim_x = ((cols - (cols % 32)) / 32) + 1;
	unsigned int grid_dim_y = ((rows - (rows % 32)) / 32) + 1;


	dim3 num_blocks(grid_dim_x, grid_dim_y, 1);
	dim3 threads_per_block(block_dim_xy, block_dim_xy, 1);


	find_labels_kernel << <num_blocks, threads_per_block >>> (src, labels, row_vals, col_vals, sector_LUT, density, k_step);
	hipDeviceSynchronize();


}