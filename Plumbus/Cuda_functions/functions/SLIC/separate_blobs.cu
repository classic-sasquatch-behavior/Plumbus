#include "hip/hip_runtime.h"
#include"../../../cuda_includes.h"
#include"../../../config.h"



__global__ void linear_flow_kernel(iptr src, iptr temp, iptr id_LUT, int N, int* change) {
	get_dims_ids_and_check_bounds
	int self_label = src(row, col);
	int self_temp_val = temp(row, col);
	int greatest_temp_val = self_temp_val;

	for_each_immediate_neighbor (
		int neighbor_label = src(neighbor_row, neighbor_col);
		if (self_label == neighbor_label) {
			int neighbor_temp_val = temp(neighbor_row, neighbor_col);
			if (neighbor_temp_val > greatest_temp_val) {
				greatest_temp_val = neighbor_temp_val;
			}
		}
	) //end for_each_immediate_neighbor

	if (greatest_temp_val != self_temp_val) {
		change[0] = 1;
	}
}



__global__ void make_coords_to_ids_LUT_kernel(iptr src) {
	get_dims_ids_and_check_bounds
	src(row, col) = id;
}



void separate_blobs_launch(gMat& labels) {
	alias_input(labels);
	get_structure_from_mat;
	make_2d_kernel_from_structure;

	cv::cuda::GpuMat temp_labels = labels;
	cv::cuda::GpuMat id_LUT(labels.size(), labels.type());
	make_coords_to_ids_LUT_kernel << < num_blocks, threads_per_block >> > (id_LUT);
	cusyncerr(make_coords_to_ids_LUT_kernel);

	int change = 0;
	int* h_flag = &change;
	int* d_flag;
	hipMalloc(&d_flag, sizeof(int));
	bool converged = false;
	int DEBUG_times_run = 0;

	while (!converged) {
		std::cout << "separate_blobs times run: " << DEBUG_times_run << std::endl;
		hipMemcpy(d_flag, h_flag, sizeof(int), hipMemcpyHostToDevice);
		linear_flow_kernel <<<num_blocks, threads_per_block>>> (labels, temp_labels, id_LUT, N, d_flag);
		cusyncerr(linear_flow_in_separate_blobs);
		hipMemcpy(h_flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);

		if (change == 0) {
			converged = true;
		}
		change = 0;
		DEBUG_times_run++;
	}
	hipFree(d_flag);

	labels = temp_labels;
}