#include "hip/hip_runtime.h"
#include"../../cuda_includes.h"








__global__ void find_borders_kernel(int width, int height, uchar* d_src, int src_row_step, int* d_focus, int* d_target, int src_elm_size) { 

	int focus_x = blockIdx.x;
	int focus_y = blockIdx.y;

	int target_x_mod = (threadIdx.x % 3) - 1;
	int target_y_mod = ((threadIdx.x - target_x_mod) / 3) - 1;

	int target_x = focus_x + target_x_mod;
	int target_y = focus_y + target_y_mod;

	if (focus_x >= width || focus_y >= height || target_x >= width || target_y >= height || target_x < 0 || target_y < 0) { return; }

	int write_position = (((blockIdx.x + (gridDim.x * blockIdx.y)) * 9) + threadIdx.x); //position within output array, which is linear
	int focus_position = (focus_y * src_row_step)+(focus_x * src_elm_size); //position within input mat, which is width*height
	int target_position = (target_y * src_row_step)+(target_x * src_elm_size); //position within input mat, which is width*height

	int focus_val = d_src[focus_position];
	int target_val = d_src[target_position];

	if (focus_val != target_val) {
		d_focus[write_position] = focus_val;
		d_target[write_position] = target_val;
	}
	else {
		d_focus[write_position] = -1;
		d_target[write_position] = -1;
	}
}






struct make_pair : public thrust::binary_function<int, int, thrust::pair<int, int>> {
	__host__ __device__
		thrust::pair<int, int> operator()(int x, int y) { return thrust::make_pair(x, y); }
};


std::vector<thrust::pair<int, int>> find_borders_launch(int src_width, int src_height, cv::cuda::GpuMat& d_src) { //just make it return a thrust::host_vector

	thrust::device_vector<int> d_focus_result(src_width * src_height * 9);
	thrust::device_vector<int> d_target_result(src_width * src_height * 9);

	int* d_focus = thrust::raw_pointer_cast(d_focus_result.data());
	int* d_target = thrust::raw_pointer_cast(d_target_result.data());

	dim3 num_blocks = {uint(src_width + 1), uint(src_height + 1)};
	dim3 threads_per_block = {9, 1, 1};
	int substep_size = sizeof(int);

	find_borders_kernel <<<num_blocks, threads_per_block>>> (src_width, src_height, d_src.data, d_src.step, d_focus, d_target, substep_size);


	//SOLVED: USE TUPLES AND A ZIP ITERATOR, NOT PAIR AND WHATEVER THIS SHIT IS



	thrust::device_vector<thrust::pair<int, int>> d_unique(d_focus_result.size());


	thrust::transform(d_focus_result.begin(), d_focus_result.end(), d_target_result.begin(), d_unique.begin(), make_pair() ); //seems to do its job just fine

	thrust::unique(thrust::device, d_unique.begin(), d_unique.end());

	std::vector<thrust::pair<int, int>> output(d_unique.size());
	thrust::copy(d_unique.begin(), d_unique.end(), output.begin() );
	return output;
}